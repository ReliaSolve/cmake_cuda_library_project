#include "hip/hip_runtime.h"
/*
 * Copyright 2021 ReliaSolve, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <algorithm>
#include "cuda_kernels.h"
#include "hip/hip_math_constants.h"

extern "C" {

  //----------------------------------------------------------------------
  // Definitions and routines needed by all functions below.
  //----------------------------------------------------------------------

  static FLOATPIXEL* g_cuda_buffer_float = nullptr;

  static unsigned		g_cuda_nx = 0;
  static unsigned		g_cuda_ny = 0;

  // For the GPU code, block size and number of kernels to run to cover a whole grid.
  // Initialized once in ensure_cuda_ready();
  static dim3         g_threads;      // 16x16x1
  static dim3         g_grid;         // Computed to cover array (slightly larger than array)

  // Open the CUDA device and get a context.  Also allocate buffers of
  // appropriate size.  Do this allocation only when the size of the buffer
  // allocated is different from the newly-requested size.  Return false
  // if we cannot get one.  This function can be called every time a
  // CUDA-using function is called, but it only does the device opening
  // and image-buffer allocation once.
  static const char *ensure_cuda_ready(unsigned nx, unsigned ny)
  {
    static bool initialized = false;	// Have we initialized yet?
    if (!initialized) {
      // Whether this works or not, we'll be initialized.
      initialized = true;

      // Open the first CUDA device in the system and create a context.
      if (hipSetDevice(0) != hipSuccess) {
        return "ensure_cuda_ready(): Could not set device.";
      }
      g_cuda_nx = nx;
      g_cuda_ny = ny;
    }

    // Allocate buffer to be used on the GPU.  It will be copied from and to host memory.
    // If we've been given a different-sized buffer, deallocate the old buffers.
    // Don't do this if we had an empty size before.
    if ( (g_cuda_nx || g_cuda_ny) && ((nx != g_cuda_nx) || (ny != g_cuda_ny)) ) {
      hipFree(g_cuda_buffer_float);
    }

    unsigned int numFloatBytes = nx * ny * sizeof(FLOATPIXEL);
    if (g_cuda_buffer_float == NULL) {
      if (hipMalloc((void**)&g_cuda_buffer_float, numFloatBytes) != hipSuccess) {
        return "ensure_cuda_ready(): Could not allocate memory";
      }
      if (g_cuda_buffer_float == NULL) {
        return "ensure_cuda_ready() : Buffer is NULL pointer.";
      }
    }

    // Return true if we are okay.
    return "";
  }

  // CUDA kernel to sum the results for all points into the image.
  static __global__ void cuda_compute_point_sum_kernel(
    const POINT* points, int numPoints, FLOATPIXEL* image, int nx, int ny)
  {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    // Only compute values for pixels that inside the image.
    if ((x < nx) && (y < ny)) {
      for (int i = 0; i < numPoints; i++) {
        const POINT& p = points[i];
        float R_squared = (x-p[0])*(x-p[0]) + (y-p[1])*(y-p[1]);
        // Avoid numerical instability
        if (R_squared == 0) {
          image[x + y * nx] += 1e10;
        } else {
          image[x + y * nx] += FLOATPIXEL(1) / R_squared;
        }
      }
    }
  }

  const char* cuda_compute_point_sum(
    const POINT* points, int numPoints, FLOATPIXEL* image, int nx, int ny)
  {
    // Check the parameters.
    if (!points) { return "cuda_compute_point_sum(): Null points"; }
    if (!image) { return "cuda_compute_point_sum(): Null image"; }
    if (numPoints <= 0) { return "cuda_compute_point_sum(): numPoints <= 0"; }
    if (nx <= 0) { return "cuda_compute_point_sum(): nx <= 0"; }
    if (ny <= 0) { return "cuda_compute_point_sum(): ny <= 0"; }

    // Make sure we can initialize CUDA.  This also allocates the global
    // input buffer that we'll copy data from the host into.
    const char* ret = ensure_cuda_ready(nx, ny);
    if (strlen(ret) != 0) { return ret; }

    // Zero the GPU image that we will sum into.
    // Copy the input image from host memory into the GPU buffers.  Don't re-copy the same
    // image that we sent last time.
    size_t floatSize = nx * ny * sizeof(FLOATPIXEL);
    if (hipMemset(g_cuda_buffer_float, 0, floatSize) != hipSuccess) {
      return "cuda_compute_point_sum(): Could not zero CUDA image";
    }

    // Allocate a GPU buffer for the points and copy the points into it.
    POINT* points_gpu = nullptr;
    size_t pointsSize = numPoints * sizeof(POINT);
    if (hipMalloc((void**)&points_gpu, pointsSize) != hipSuccess) {
      return "cuda_compute_point_sum(): Could not allocate memory";
    }
    if (hipMemcpy(points_gpu, points, pointsSize, hipMemcpyHostToDevice) != hipSuccess) {
      return "cuda_compute_point_sum(): Could not copy points to GPU";
    }

    // Set up enough threads (and enough blocks of threads) to at least
    // cover the size of the array.  We use a thread block size of 16x16
    // because that's what the example matrixMul code from nVidia does.
    g_threads.x = 16;
    g_threads.y = 16;
    g_threads.z = 1;
    g_grid.x = ((g_cuda_nx - 1) / g_threads.x) + 1;
    g_grid.y = ((g_cuda_ny - 1) / g_threads.y) + 1;
    g_grid.z = 1;

    // Call the CUDA kernel to make the output image,
    // Synchronize the threads when we are done so we know that they finish
    // before we copy the memory back.
    cuda_compute_point_sum_kernel << < g_grid, g_threads >> > (
      points_gpu, numPoints, g_cuda_buffer_float, nx, ny);
    if (hipDeviceSynchronize() != hipSuccess) {
      return "cuda_compute_point_sum(): Could not synchronize threads";
    }

    // Copy the buffer back from the GPU to host memory.
    if (hipMemcpy(image, g_cuda_buffer_float, floatSize, hipMemcpyDeviceToHost) != hipSuccess) {
      return "cuda_compute_point_sum(): Could not copy image back to host";
    }

    // We're done with the points buffer now.
    hipFree(points_gpu);

    // Everything worked!
    return "";
  }

}